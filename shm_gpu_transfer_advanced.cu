#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define TOTAL_SIZE (1024L * 1024L * 1024L)  // 1GB
#define BLOCK_SIZE (256L * 1024L * 1024L)   // 256MB
#define NUM_BLOCKS (TOTAL_SIZE / BLOCK_SIZE)
#define ITERATIONS 10

void checkCuda(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

int main() {
    printf("=== SHM → GPU0 → GPU1 Transfer Demo (Advanced) ===\n");

    // 1. 创建共享内存
    int shm_fd = shm_open("/my_shm", O_CREAT | O_RDWR, 0666);
    ftruncate(shm_fd, TOTAL_SIZE);
    void* shm_ptr = mmap(0, TOTAL_SIZE, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);

    // 初始化共享内存内容
    for (size_t i = 0; i < TOTAL_SIZE; ++i) {
        ((char*)shm_ptr)[i] = i % 256;
    }

    // 注册为 pinned memory
    checkCuda(hipHostRegister(shm_ptr, TOTAL_SIZE, hipHostRegisterPortable), "Register SHM");

    // 分配 GPU 内存
    hipSetDevice(0);
    void* gpu0_ptr;
    checkCuda(hipMalloc(&gpu0_ptr, TOTAL_SIZE), "GPU0 malloc");

    hipSetDevice(1);
    void* gpu1_ptr;
    checkCuda(hipMalloc(&gpu1_ptr, TOTAL_SIZE), "GPU1 malloc");

    // 检查 P2P 支持
    int canAccessPeer = 0;
    hipDeviceCanAccessPeer(&canAccessPeer, 1, 0);
    if (canAccessPeer) {
        hipSetDevice(1);
        hipDeviceEnablePeerAccess(0, 0);
        printf("P2P enabled between GPU0 and GPU1.\n");
    } else {
        printf("P2P not supported. Using CPU fallback.\n");
    }

    float total_shm_to_gpu0 = 0.0f;
    float total_gpu0_to_gpu1 = 0.0f;

    for (int iter = 0; iter < ITERATIONS; ++iter) {
        hipEvent_t start1, stop1, start2, stop2;
        hipEventCreate(&start1); hipEventCreate(&stop1);
        hipEventCreate(&start2); hipEventCreate(&stop2);

        hipEventRecord(start1, 0);
        for (int b = 0; b < NUM_BLOCKS; ++b) {
            void* shm_block = (char*)shm_ptr + b * BLOCK_SIZE;
            void* gpu0_block = (char*)gpu0_ptr + b * BLOCK_SIZE;
            checkCuda(hipMemcpy(gpu0_block, shm_block, BLOCK_SIZE, hipMemcpyHostToDevice), "Memcpy SHM → GPU0");
        }
        hipEventRecord(stop1, 0);
        hipEventSynchronize(stop1);
        float t1 = 0;
        hipEventElapsedTime(&t1, start1, stop1);
        total_shm_to_gpu0 += t1;

        hipEventRecord(start2, 0);
        for (int b = 0; b < NUM_BLOCKS; ++b) {
            void* gpu0_block = (char*)gpu0_ptr + b * BLOCK_SIZE;
            void* gpu1_block = (char*)gpu1_ptr + b * BLOCK_SIZE;

            if (canAccessPeer) {
                checkCuda(hipMemcpyPeer(gpu1_block, 1, gpu0_block, 0, BLOCK_SIZE), "MemcpyPeer GPU0 → GPU1");
            } else {
                void* cpu_buf = malloc(BLOCK_SIZE);
                hipMemcpy(cpu_buf, gpu0_block, BLOCK_SIZE, hipMemcpyDeviceToHost);
                hipMemcpy(gpu1_block, cpu_buf, BLOCK_SIZE, hipMemcpyHostToDevice);
                free(cpu_buf);
            }
        }
        hipEventRecord(stop2, 0);
        hipEventSynchronize(stop2);
        float t2 = 0;
        hipEventElapsedTime(&t2, start2, stop2);
        total_gpu0_to_gpu1 += t2;

        hipEventDestroy(start1); hipEventDestroy(stop1);
        hipEventDestroy(start2); hipEventDestroy(stop2);
    }

    float avg_shm_to_gpu0 = total_shm_to_gpu0 / ITERATIONS;
    float avg_gpu0_to_gpu1 = total_gpu0_to_gpu1 / ITERATIONS;
    float total_avg = avg_shm_to_gpu0 + avg_gpu0_to_gpu1;
    float bandwidth = 1.0f / (total_avg / 1000.0f);  // GB/s

    printf("\n=== Performance Summary ===\n");
    printf("Average SHM → GPU0 time: %.3f ms\n", avg_shm_to_gpu0);
    printf("Average GPU0 → GPU1 time: %.3f ms\n", avg_gpu0_to_gpu1);
    printf("Total average transfer time: %.3f ms\n", total_avg);
    printf("Effective bandwidth: %.2f GB/s\n", bandwidth);

    // 清理资源
    hipFree(gpu0_ptr);
    hipFree(gpu1_ptr);
    hipHostUnregister(shm_ptr);
    munmap(shm_ptr, TOTAL_SIZE);
    shm_unlink("/my_shm");

    return 0;
}

