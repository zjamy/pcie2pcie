#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define SHM_NAME "/my_shm"
//#define SHM_SIZE 1024 * 1024  // 1MB 示例数据
#define SHM_SIZE (1024L * 1024L * 1024L)  // 1GB


// 计时辅助函数（CPU）
double get_time_ms() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000.0 + tv.tv_usec / 1000.0;
}

// CUDA 错误检查
void checkCuda(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

int main() {
    printf("=== SHM → GPU0 → GPU1 Transfer Demo ===\n");

    // 1. 创建共享内存
    int shm_fd = shm_open(SHM_NAME, O_CREAT | O_RDWR, 0666);
    ftruncate(shm_fd, SHM_SIZE);
    void* shm_ptr = mmap(0, SHM_SIZE, PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);

    // 初始化共享内存内容
    for (int i = 0; i < SHM_SIZE; ++i) {
        ((char*)shm_ptr)[i] = i % 256;
    }

    // 2. 注册共享内存为 CUDA pinned memory
    checkCuda(hipHostRegister(shm_ptr, SHM_SIZE, hipHostRegisterPortable), "Register SHM");

    // 3. 分配 GPU0 和 GPU1 的内存
    hipSetDevice(0);
    void* gpu0_ptr;
    checkCuda(hipMalloc(&gpu0_ptr, SHM_SIZE), "GPU0 malloc");

    hipSetDevice(1);
    void* gpu1_ptr;
    checkCuda(hipMalloc(&gpu1_ptr, SHM_SIZE), "GPU1 malloc");

    // 4. 拷贝 SHM → GPU0
    hipSetDevice(0);
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1, 0);

    checkCuda(hipMemcpy(gpu0_ptr, shm_ptr, SHM_SIZE, hipMemcpyHostToDevice), "Memcpy SHM to GPU0");

    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);
    float time_shm_to_gpu0 = 0;
    hipEventElapsedTime(&time_shm_to_gpu0, start1, stop1);
    printf("SHM → GPU0 time: %.3f ms\n", time_shm_to_gpu0);

    // 5. GPU0 → GPU1
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2, 0);

    int canAccessPeer = 0;
    hipDeviceCanAccessPeer(&canAccessPeer, 1, 0);
    if (canAccessPeer) {
        hipSetDevice(1);
        hipDeviceEnablePeerAccess(0, 0);
        checkCuda(hipMemcpyPeer(gpu1_ptr, 1, gpu0_ptr, 0, SHM_SIZE), "MemcpyPeer GPU0 to GPU1");
        printf("P2P transfer successful.\n");
    } else {
        printf("P2P not supported. Falling back to CPU copy.\n");
        void* cpu_buffer = malloc(SHM_SIZE);
        hipMemcpy(cpu_buffer, gpu0_ptr, SHM_SIZE, hipMemcpyDeviceToHost);
        hipMemcpy(gpu1_ptr, cpu_buffer, SHM_SIZE, hipMemcpyHostToDevice);
        free(cpu_buffer);
    }

    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    float time_gpu0_to_gpu1 = 0;
    hipEventElapsedTime(&time_gpu0_to_gpu1, start2, stop2);
    printf("GPU0 → GPU1 time: %.3f ms\n", time_gpu0_to_gpu1);

    // 6. 总结
    printf("Total transfer time: %.3f ms\n", time_shm_to_gpu0 + time_gpu0_to_gpu1);
    printf("Effective bandwidth: %.2f MB/s\n", SHM_SIZE / 1024.0 / (time_shm_to_gpu0 + time_gpu0_to_gpu1));

    // 7. 清理资源
    hipFree(gpu0_ptr);
    hipFree(gpu1_ptr);
    hipHostUnregister(shm_ptr);
    munmap(shm_ptr, SHM_SIZE);
    shm_unlink(SHM_NAME);

    return 0;
}

